#include <hip/hip_runtime.h>
#include <cstdlib>


template<int SHARED_MEM_SIZE>
__global__ void reduce_kernel_v1(float *input, float *output) {
  __shared__ float shmem[SHARED_MEM_SIZE];

  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int i = threadIdx.x;
  
  shmem[threadIdx.x] = input[tid];
  __syncthreads();

  for (int stride = blockDim.x / 2; stride >= 1; stride >>= 1) {
    if (threadIdx.x < stride) {
      shmem[i] += shmem[i + stride];
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    output[blockIdx.x] = shmem[0];
  }
}

void reduce_v1(float *input, size_t input_count, float *output) {
  size_t input_size = input_count * sizeof(float);
  const int THREAD_COUNT = 32;
  const int BLOCK_COUNT = (input_count + THREAD_COUNT - 1) / THREAD_COUNT;

  size_t output_size = BLOCK_COUNT * sizeof(float);

  float *input_dev = nullptr;
  auto err = hipMalloc(&input_dev, input_size);
  hipMemcpy(input_dev, input, input_size,
             hipMemcpyKind::hipMemcpyHostToDevice);

  float *output_dev = nullptr;
  err = hipMalloc(&output_dev, output_size);
  hipMemcpy(output_dev, output, output_size,
             hipMemcpyKind::hipMemcpyHostToDevice);
  float *output_host = (float *)std::malloc(output_size);

  reduce_kernel_v1<THREAD_COUNT><<<BLOCK_COUNT, THREAD_COUNT>>>(input_dev, output_dev);

  hipMemcpy(output_host, output_dev, output_size,
             hipMemcpyKind::hipMemcpyDeviceToHost);

  float sum = 0.0f;
  for (int i = 0; i < BLOCK_COUNT; ++i) {
    sum += output_host[i];
  }

  *output = sum;
}