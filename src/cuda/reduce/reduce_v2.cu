#include "hip/hip_runtime.h"
#include "benchmark.cuh"
#include "util.h"
#include <cstddef>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <vector>

template <int SHARED_MEM_SIZE, int COARSE_FACTOR>
__global__ void reduce_kernel_v2(float *input, float *output) {
  __shared__ float shmem[SHARED_MEM_SIZE];

  const int segment = COARSE_FACTOR * blockDim.x * blockIdx.x;
  const int tx = threadIdx.x;
  const int i = segment + tx;
  float sum = input[i];

#pragma unroll
  for (int tile = 1; tile < COARSE_FACTOR; ++tile) {
    sum += input[i + tile * SHARED_MEM_SIZE];
  }
  shmem[tx] = sum;

  for (int stride = blockDim.x / 2; stride >= 1; stride >>= 1) {
    __syncthreads();
    if (tx < stride) {
      shmem[tx] += shmem[tx + stride];
    }
  }

  if (tx == 0) {
    output[blockIdx.x] = shmem[0];
  }
}

void reduce_v2(float *input, size_t input_count, float *output) {
  size_t input_size = input_count * sizeof(float);
  const int THREAD_COUNT = 32;
  const int COARSE_FACTOR = 4;
  const int BLOCK_COUNT = (input_count + THREAD_COUNT * COARSE_FACTOR - 1) /
                          (THREAD_COUNT * COARSE_FACTOR);

  size_t output_size = BLOCK_COUNT * sizeof(float);

  float *input_dev = nullptr;
  auto err = hipMalloc(&input_dev, input_size);
  hipMemcpy(input_dev, input, input_size,
             hipMemcpyKind::hipMemcpyHostToDevice);

  float *output_dev = nullptr;
  err = hipMalloc(&output_dev, output_size);
  hipMemcpy(output_dev, output, output_size,
             hipMemcpyKind::hipMemcpyHostToDevice);
  float *output_host = (float *)std::malloc(output_size);

  reduce_kernel_v2<THREAD_COUNT, COARSE_FACTOR>
      <<<BLOCK_COUNT, THREAD_COUNT>>>(input_dev, output_dev);

  hipMemcpy(output_host, output_dev, output_size,
             hipMemcpyKind::hipMemcpyDeviceToHost);

  float sum = 0.0f;
  for (int i = 0; i < BLOCK_COUNT; ++i) {
    sum += output_host[i];
  }

  *output = sum;
}

void reduce_v2_benchmark() {
  const size_t count = 32 * 1024 * 1024;
  const size_t input_size = count * sizeof(float);
  const int repeat = 1;

  std::vector<float> input(count, 0.0f);
  init_random(input);
  const int THREAD_COUNT = 64;
  const int COARSE_FACTOR = 4;
  const int BLOCK_COUNT = (count + THREAD_COUNT * COARSE_FACTOR - 1) /
                          (THREAD_COUNT * COARSE_FACTOR);
  std::vector<float> output(BLOCK_COUNT, 0.0f);

  float *input_dev = nullptr;
  auto err = hipMalloc(&input_dev, input_size);
  float *output_dev = nullptr;
  err = hipMalloc(&output_dev, BLOCK_COUNT * sizeof(float));

  hipMemcpy(input_dev, input.data(), input_size,
             hipMemcpyKind::hipMemcpyHostToDevice);
  hipMemcpy(output_dev, output.data(), BLOCK_COUNT * sizeof(float),
             hipMemcpyKind::hipMemcpyHostToDevice);

  double flops = 1.0 * count;
  double bytes = 2.0 * input_size;

  benchmarkKernel("reduce_kernel_v2<32, 32>", BLOCK_COUNT, THREAD_COUNT, flops,
                  bytes, repeat, reduce_kernel_v2<THREAD_COUNT, COARSE_FACTOR>,
                  input_dev, output_dev);
}