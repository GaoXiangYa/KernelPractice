#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <hip/hip_runtime.h>

// a[m, n] * b[n, k] = c[m, k]
__global__ void matmul_native_kernel(const float *a, const float *b, float *c,
                                     int m, int n, int k) {
  const int col = blockIdx.x * blockDim.x + threadIdx.x;
  const int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row < m && col < k) {
    float sum = 0;
    for (int i = 0; i < n; ++i) {
      sum += a[row * n + i] * b[i * k + col];
    }
    c[row * k + col] = sum;
  }
}

// 使用shared memory进行优化
template <int BLOCK>
__global__ void matmul_sharedmemory_kernel(float *A, float *B, float *C, int M, int N,
                                int K) {
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int bx = blockIdx.x;
  const int by = blockIdx.y;

  float *begin_a = A + by * BLOCK * K;
  float *begin_b = B + bx * BLOCK;
  float *end_a = begin_a + K;

  float sum = 0.0f;
  for (auto a_ptr = begin_a, b_ptr = begin_b; a_ptr < end_a;
       a_ptr += BLOCK, b_ptr += BLOCK * N) {
    __shared__ float shared_a[BLOCK][BLOCK];
    __shared__ float shared_b[BLOCK][BLOCK];
    shared_a[ty][tx] = a_ptr[ty * K + tx];
    shared_b[ty][tx] = b_ptr[ty * N + tx];
    __syncthreads();

#pragma unroll
    for (int k = 0; k < BLOCK; ++k) {
      sum += shared_a[ty][k] * shared_b[k][tx];
    }

    __syncthreads();
  }

  C[(BLOCK * by + ty) * N + BLOCK * bx + tx] = sum;
}

void matmul_native(const float *a, const float *b, float *c, int m, int n,
                   int k) {
  float *dev_a = nullptr;
  auto err = hipMalloc(&dev_a, m * n * sizeof(float));
  err = hipMemcpy(dev_a, a, m * n * sizeof(float), hipMemcpyHostToDevice);
  
  float *dev_b = nullptr;
  err = hipMalloc(&dev_b, n * k * sizeof(float));
  err = hipMemcpy(dev_b, b, n * k * sizeof(float), hipMemcpyHostToDevice);

  float *dev_c = nullptr;
  err = hipMalloc(&dev_c, m * k * sizeof(float));
  err = hipMemcpy(dev_c, c, m * k * sizeof(float), hipMemcpyHostToDevice);

  const int THREAD_COUNT = 32;
  dim3 block(THREAD_COUNT, THREAD_COUNT);
  dim3 grid((k + block.x - 1) / block.x, (m + block.y - 1) / block.y);

  matmul_native_kernel<<<grid,block>>>(dev_a, dev_b, dev_c, m, n, k);

  hipMemcpy(c, dev_c, m * k * sizeof(float), hipMemcpyDeviceToHost);
}

void matmul_sharedmemory(const float *a, const float *b, float *c, int m, int n, int k) {}
